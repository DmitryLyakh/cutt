#include "hip/hip_runtime.h"
/******************************************************************************
MIT License

Copyright (c) 2016 Antti-Pekka Hynninen
Copyright (c) 2016 Oak Ridge National Laboratory (UT-Batelle)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*******************************************************************************/
#include <hip/hip_runtime.h>
#include "CudaUtils.h"
#include "cuttkernel.h"

//
// Returns scalar tensor position. Each lane has the same p
// NOTE: c and d on inactive warps must be 1 !!
//
__device__ __forceinline__
int tensorPos(
  const int p, const int rank, const int c, const int d, const int ct,
  const int numLane=warpSize
  ) {

  int r = ((p/c) % d)*ct;
#pragma unroll
  for (int i=numLane/2;i >= 1;i/=2) {
    r += __shfl_xor(r, i);
  }
  return r;

}

//__constant__ int args[2];

#if 0
//
// Transpose when Mm and Mk don't overlap and contain only single rank
//
//  dim3 numthread(TILEDIM, TILEROWS, 1);
//  dim3 numblock((plan.volMm-1)/TILEDIM+1, (plan.volMk-1)/TILEDIM+1, plan.volMbar);
//
template <typename T>
__global__ void transposeTiledSingleRank(
  const int volMbar, const int sizeMbar,
  const int2 readVol, const int cuDimMk, const int cuDimMm,
  const TensorConvInOut* __restrict__ glMbar,
  const T* __restrict__ dataIn, T* __restrict__ dataOut) {

  // Shared memory
  __shared__ T shTile[TILEDIM][TILEDIM+1];

  const int warpLane = threadIdx.x & (warpSize - 1);
  TensorConvInOut Mbar;
  Mbar.c_in = 1;
  Mbar.d_in = 1;
  Mbar.c_out = 1;
  Mbar.d_out = 1;
  if (warpLane < sizeMbar) {
    Mbar = glMbar[warpLane];
  }

  for (int posMbar=blockIdx.z;posMbar < volMbar;posMbar += gridDim.z)
  {

    // Compute global memory positions
    int posMajorIn = ((posMbar/Mbar.c_in) % Mbar.d_in)*Mbar.ct_in;
    int posMajorOut = ((posMbar/Mbar.c_out) % Mbar.d_out)*Mbar.ct_out;
#pragma unroll
    for (int i=16;i >= 1;i/=2) {
      posMajorIn += __shfl_xor(posMajorIn, i);
      posMajorOut += __shfl_xor(posMajorOut, i);
    }

    for (int x0 = blockIdx.x * TILEDIM;x0 < readVol.x;x0 += blockDim.x*gridDim.x) {
      for (int y0 = blockIdx.y * TILEDIM;y0 < readVol.y;y0 += blockDim.y*gridDim.y) {

        int xin = x0 + threadIdx.x;
        int yin = y0 + threadIdx.y;

        int xout = x0 + threadIdx.y;
        int yout = y0 + threadIdx.x;

        int posIn = posMajorIn + xin + yin*cuDimMk;
        int posOut = posMajorOut + yout + xout*cuDimMm;

        // Read from global memory
        __syncthreads();

        // Read data into shared memory tile
    #pragma unroll
        for (int j=0;j < TILEDIM;j += TILEROWS) {
          int pos = posIn + j*cuDimMk;
          if (xin < readVol.x && yin + j < readVol.y) {
            shTile[threadIdx.y + j][threadIdx.x] = dataIn[pos];
          }
        }

        // Write to global memory
        __syncthreads();

    #pragma unroll
        for (int j=0;j < TILEDIM;j += TILEROWS) {
          int pos = posOut + j*cuDimMm;
          if (xout + j < readVol.x && yout < readVol.y) {
            dataOut[pos] = shTile[threadIdx.x][threadIdx.y + j];
          }
        }
      }
    }
  }
  
}
#endif

#if 1
//
// Transpose when Mm and Mk don't overlap and contain only single rank
//
//  dim3 numthread(TILEDIM, TILEROWS, 1);
//  dim3 numblock((plan.volMm-1)/TILEDIM+1, (plan.volMk-1)/TILEDIM+1, plan.volMbar);
//
template <typename T>
__global__ void transposeTiledSingleRank(
  const int volMbar, const int sizeMbar,
  const int2 tiledVol, const int cuDimMk, const int cuDimMm,
  const TensorConvInOut* __restrict__ glMbar,
  const T* __restrict__ dataIn, T* __restrict__ dataOut) {

  // Shared memory
  __shared__ T shTile[TILEDIM][TILEDIM+1];

  const int warpLane = threadIdx.x & (warpSize - 1);
  TensorConvInOut Mbar;
  Mbar.c_in = 1;
  Mbar.d_in = 1;
  Mbar.c_out = 1;
  Mbar.d_out = 1;
  if (warpLane < sizeMbar) {
    Mbar = glMbar[warpLane];
  }

  const int xin = blockIdx.x * TILEDIM + threadIdx.x;
  const int yin = blockIdx.y * TILEDIM + threadIdx.y;

  const int xout = blockIdx.x * TILEDIM + threadIdx.y;
  const int yout = blockIdx.y * TILEDIM + threadIdx.x;

  const unsigned int maskIny = __ballot((yin + warpLane < tiledVol.y))*(xin < tiledVol.x);
  const unsigned int maskOutx = __ballot((xout + warpLane < tiledVol.x))*(yout < tiledVol.y);

  const int posMinorIn = xin + yin*cuDimMk;
  const int posMinorOut = yout + xout*cuDimMm;
  const int posInAdd = TILEROWS*cuDimMk;
  const int posOutAdd = TILEROWS*cuDimMm;

  for (int posMbar=blockIdx.z;posMbar < volMbar;posMbar += gridDim.z)
  {

    // Compute global memory positions
    int posMajorIn = ((posMbar/Mbar.c_in) % Mbar.d_in)*Mbar.ct_in;
    int posMajorOut = ((posMbar/Mbar.c_out) % Mbar.d_out)*Mbar.ct_out;
#pragma unroll
    for (int i=16;i >= 1;i/=2) {
      posMajorIn += __shfl_xor(posMajorIn, i);
      posMajorOut += __shfl_xor(posMajorOut, i);
    }
    int posIn = posMajorIn + posMinorIn;
    int posOut = posMajorOut + posMinorOut;

    // Read from global memory
    __syncthreads();

    // Read data into shared memory tile
#pragma unroll
    for (int j=0;j < TILEDIM;j += TILEROWS) {
      // int pos = posIn + j*cuDimMk;
      // if (xin < readVol.x && yin + j < readVol.y) {
      if ((maskIny & (1 << j)) != 0) {
        shTile[threadIdx.y + j][threadIdx.x] = dataIn[posIn];
      }
      posIn += posInAdd;
    }

    // Write to global memory
    __syncthreads();

#pragma unroll
    for (int j=0;j < TILEDIM;j += TILEROWS) {
      // int pos = posOut + j*cuDimMm;
      // if (xout + j < readVol.x && yout < readVol.y) {
      if ((maskOutx & (1 << j)) != 0 ) {
        dataOut[posOut] = shTile[threadIdx.x][threadIdx.y + j];
      }
      posOut += posOutAdd;
    }

  }
  
}
#endif

//
// General transpose. Thread block loads plan.volMmk number of elements
//
template <typename T, int numRegStorage>
__global__ void transposeGeneral(
  const int volMm, const int volMk, const int volMmk, const int volMbar,
  const int sizeMmk, const int sizeMbar,
  const TensorConvInOut* __restrict__ gl_Mmk,
  const TensorConvInOut* __restrict__ gl_Mbar,
  const TensorConv* __restrict__ gl_Msh,
  const T* __restrict__ dataIn, T* __restrict__ dataOut) {

  // Shared memory. volMmk elements
  extern __shared__ char shBuffer_char[];
  T* shBuffer = (T *)shBuffer_char;

  const int warpLane = threadIdx.x & (warpSize - 1);

  TensorConvInOut Mmk;
  Mmk.c_in = 1;
  Mmk.d_in = 1;
  Mmk.c_out = 1;
  Mmk.d_out = 1;
  if (warpLane < sizeMmk) {
    Mmk = gl_Mmk[warpLane];
  }
  TensorConv Msh;
  Msh.c = 1;
  Msh.d = 1;
  if (warpLane < sizeMmk) {
    Msh = gl_Msh[warpLane];
  }

  // Pre-compute tensor positions in Mmk
  // 3*numRegStorage registers
  int posMmkIn[numRegStorage];
  int posMmkOut[numRegStorage];
  int posSh[numRegStorage];
#pragma unroll
  for (int j=0;j < numRegStorage;j++) {
    posMmkIn[j] = 0;
    posMmkOut[j] = 0;
    posSh[j] = 0;
  }
  for (int i=0;i < sizeMmk;i++) {
#pragma unroll
    for (int j=0;j < numRegStorage;j++) {
      posMmkIn[j]  += (((threadIdx.x + j*blockDim.x)/__shfl(Mmk.c_in,i)) % __shfl(Mmk.d_in,i))*__shfl(Mmk.ct_in,i);
      posMmkOut[j] += (((threadIdx.x + j*blockDim.x)/__shfl(Mmk.c_out,i)) % __shfl(Mmk.d_out,i))*__shfl(Mmk.ct_out,i);
      posSh[j] += (((threadIdx.x + j*blockDim.x)/__shfl(Msh.c,i)) % __shfl(Msh.d,i))*__shfl(Msh.ct,i);
    }
  }

  // 6 registers
  TensorConvInOut Mbar;
  Mbar.c_in = 1;
  Mbar.d_in = 1;
  Mbar.c_out = 1;
  Mbar.d_out = 1;
  if (warpLane < sizeMbar) {
    Mbar = gl_Mbar[warpLane];
  }

  for (int posMbar=blockIdx.x;posMbar < volMbar;posMbar += gridDim.x)
  {

    int posMbarOut = ((posMbar/Mbar.c_out) % Mbar.d_out)*Mbar.ct_out;
#pragma unroll
    for (int i=16;i >= 1;i/=2) {
      posMbarOut += __shfl_xor(posMbarOut, i);
    }

    // Read from global memory
    // int posMbarIn = tensorPos(posMbar, sizeMbar, Mbar.c_in, Mbar.d_in, Mbar.ct_in);
    int posMbarIn = ((posMbar/Mbar.c_in) % Mbar.d_in)*Mbar.ct_in;
#pragma unroll
    for (int i=16;i >= 1;i/=2) {
      posMbarIn += __shfl_xor(posMbarIn, i);
    }

    __syncthreads();

#pragma unroll
    for (int j=0;j < numRegStorage;j++) {
      int posMmk = threadIdx.x + j*blockDim.x;
      int posIn = posMbarIn + posMmkIn[j];
      if (posMmk < volMmk) shBuffer[posMmk] = dataIn[posIn];
    }

    // Write to global memory
      // int posMbarOut = tensorPos(posMbar, sizeMbar, Mbar.c_out, Mbar.d_out, Mbar.ct_out);
//       int posMbarOut = ((posMbar/Mbar.c_out) % Mbar.d_out)*Mbar.ct_out;
// #pragma unroll
//       for (int i=16;i >= 1;i/=2) {
//         posMbarOut += __shfl_xor(posMbarOut, i);
//       }

    __syncthreads();

#pragma unroll
    for (int j=0;j < numRegStorage;j++) {
      int posMmk = threadIdx.x + j*blockDim.x;
      int posOut = posMbarOut + posMmkOut[j];
      if (posMmk < volMmk) dataOut[posOut] = shBuffer[posSh[j]];
    }


  }
  
}

//
// Transpose when the lead dimension is the same, e.g. (1, 2, 3) -> (1, 3, 2)
//
//  dim3 numthread(TILEDIM, TILEROWS, 1);
//  dim3 numblock((plan.volMm-1)/TILEDIM+1, (plan.volMk-1)/TILEDIM+1, plan.volMbar);
//
template <typename T>
__global__ void transposeTiledLeadVolSame(
  const int volMbar, const int sizeMbar,
  const int cuDimMk, const int cuDimMm,
  const int2 tiledVol,
  const TensorConvInOut* __restrict__ gl_Mbar,
  const T* __restrict__ dataIn, T* __restrict__ dataOut) {

  const int warpLane = threadIdx.x & (warpSize - 1);
  TensorConvInOut Mbar;
  Mbar.c_in = 1;
  Mbar.d_in = 1;
  Mbar.c_out = 1;
  Mbar.d_out = 1;
  if (warpLane < sizeMbar) {
    Mbar = gl_Mbar[warpLane];
  }

  // int* dimMmkIn  = &transposeArg[0];

  const int x = blockIdx.x * TILEDIM + threadIdx.x;
  const int y = blockIdx.y * TILEDIM + threadIdx.y;

  for (int posMbar=blockIdx.z;posMbar < volMbar;posMbar += gridDim.z)
  {

    // Variables where values are stored
    T val[TILEDIM/TILEROWS];

    // Read global memory
    {
      int pos0 = tensorPos(posMbar, sizeMbar, Mbar.c_in, Mbar.d_in, Mbar.ct_in);
      pos0 += x + y*cuDimMk;

#pragma unroll
      for (int j=0;j < TILEDIM;j += TILEROWS) {
        int pos  = pos0  + j*cuDimMk;
        if ((x < tiledVol.x) && (y + j < tiledVol.y)) {
        // if ((x < dimMmkIn[0]) && (y + j < dimMmkIn[1])) {
          val[j/TILEROWS] = dataIn[pos];
        }
      }
    }

    // Write global memory
    {
      int pos0 = tensorPos(posMbar, sizeMbar, Mbar.c_out, Mbar.d_out, Mbar.ct_out);
      pos0 += x + y*cuDimMm;

#pragma unroll
      for (int j=0;j < TILEDIM;j += TILEROWS) {
        int pos = pos0 + j*cuDimMm;
        if ((x < tiledVol.x) && (y + j < tiledVol.y)) {
        // if ((x < dimMmkIn[0]) && (y + j < dimMmkIn[1])) {
          dataOut[pos] = val[j/TILEROWS];
        }
      }
    }

  }
  
}

//######################################################################################
//######################################################################################
//######################################################################################

//
// Sets shared memory bank configuration for all kernels. Needs to be called once per device.
//
void cuttKernelSetSharedMemConfig() {  
#define CALL(NREG) cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeGeneral<float), NREG>, hipSharedMemBankSizeFourByte ))
    CALL(1);
    CALL(2);
    CALL(3);
    CALL(4);
    CALL(5);
    CALL(6);
    CALL(7);
    CALL(8);
#undef CALL
#define CALL(NREG) cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeGeneral<double), NREG>, hipSharedMemBankSizeEightByte ))
    CALL(1);
    CALL(2);
    CALL(3);
    CALL(4);
    CALL(5);
    CALL(6);
    CALL(7);
    CALL(8);
#undef CALL

  cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeTiledSingleRank<float>), hipSharedMemBankSizeFourByte));
  cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeTiledLeadVolSame<float>), hipSharedMemBankSizeFourByte));

  cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeTiledSingleRank<double>), hipSharedMemBankSizeEightByte));
  cudaCheck(hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(transposeTiledLeadVolSame<double>), hipSharedMemBankSizeEightByte));
}

//
// Returns the maximum number of active blocks per SM
//
int getNumActiveBlock(int method, int sizeofType, LaunchConfig& lc) {
  int numActiveBlock;
  int numthread = lc.numthread.x * lc.numthread.y * lc.numthread.z;
  switch(method) {
    case cuttPlan_t::General:
    {
    #define CALL(TYPE, NREG) \
      hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock, \
        transposeGeneral<TYPE, NREG>, numthread, lc.shmemsize)
      switch(lc.numRegStorage) {
        case 1:
        if (sizeofType == 4) CALL(float,  1);
        if (sizeofType == 8) CALL(double, 1);
        break;
        case 2:
        if (sizeofType == 4) CALL(float,  2);
        if (sizeofType == 8) CALL(double, 2);
        break;
        case 3:
        if (sizeofType == 4) CALL(float,  3);
        if (sizeofType == 8) CALL(double, 3);
        break;
        case 4:
        if (sizeofType == 4) CALL(float,  4);
        if (sizeofType == 8) CALL(double, 4);
        break;
        case 5:
        if (sizeofType == 4) CALL(float,  5);
        if (sizeofType == 8) CALL(double, 5);
        break;
        case 6:
        if (sizeofType == 4) CALL(float,  6);
        if (sizeofType == 8) CALL(double, 6);
        break;
        case 7:
        if (sizeofType == 4) CALL(float,  7);
        if (sizeofType == 8) CALL(double, 7);
        break;
        case 8:
        if (sizeofType == 4) CALL(float,  8);
        if (sizeofType == 8) CALL(double, 8);
        break;
      }
    #undef CALL
    }
    break;
    case cuttPlan_t::TiledSingleRank:
    {
      if (sizeofType == 4) {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock,
          transposeTiledSingleRank<float>, numthread, lc.shmemsize);
      } else {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock,
          transposeTiledSingleRank<double>, numthread, lc.shmemsize);
      }
    }
    break;
    case cuttPlan_t::TiledLeadVolSame:
    {
      if (sizeofType == 4) {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock,
          transposeTiledLeadVolSame<float>, numthread, lc.shmemsize);
      } else {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock,
          transposeTiledLeadVolSame<double>, numthread, lc.shmemsize);
      }
    }
    break;
  }

  return numActiveBlock;
}

//
// Sets up kernel launch configuration
//
// Returns the number of active blocks per SM that can be achieved on the General kernel
// NOTE: Returns 0 when kernel execution is not possible
//
// Sets:
// lc.numthread
// lc.numblock
// lc.shmemsize
// lc.numRegStorage  (for General method)
//
int cuttKernelLaunchConfiguration(int method, int sizeofType, TensorSplit& ts, hipDeviceProp_t& prop,
  LaunchConfig& lc) {

  switch(method) {
    case cuttPlan_t::General:
    {
      // Amount of shared memory required
      lc.shmemsize = ts.volMmk*sizeofType;

      // Check that we're not using too much shared memory per block
      if (lc.shmemsize > prop.sharedMemPerBlock) return 0;

      // Min and max number of threads we can use
      int minNumthread = ((ts.volMmk - 1)/(prop.warpSize*MAX_REG_STORAGE) + 1)*prop.warpSize;
      int maxNumthread = ((ts.volMmk - 1)/(prop.warpSize) + 1)*prop.warpSize;      
      if (minNumthread > prop.maxThreadsPerBlock) return 0;
      maxNumthread = min(prop.maxThreadsPerBlock, maxNumthread);

      // Min and max number of register storage we can use
      // int minNumRegStorage = (ts.volMmk - 1)/maxNumthread + 1;
      // int maxNumRegStorage = (ts.volMmk - 1)/minNumthread + 1;

      if (maxNumthread <= 256) {
        lc.numRegStorage = (ts.volMmk - 1)/128 + 1;
      } else {
        lc.numRegStorage = 6;
      }

/*
      int bestNumActiveBlock = 0;
      int bestNumRegStorage = 0;

      for (lc.numRegStorage=minNumRegStorage;lc.numRegStorage <= maxNumRegStorage;lc.numRegStorage++) {
        lc.numthread.x = ((ts.volMmk - 1)/(prop.warpSize*lc.numRegStorage) + 1)*prop.warpSize;
        lc.numthread.x = min(prop.maxThreadsPerBlock, lc.numthread.x);
        lc.numthread.y = 1;
        lc.numthread.z = 1;
        lc.numblock.x = max(1, ts.volMbar);
        lc.numblock.x = min(256, lc.numblock.x);
        lc.numblock.y = 1;
        lc.numblock.z = 1;

        int numActiveBlock = getNumActiveBlock(method, sizeofType, lc);
        if (numActiveBlock > bestNumActiveBlock) {
          bestNumActiveBlock = numActiveBlock;
          bestNumRegStorage = lc.numRegStorage;
        }

        // lc.numRegStorage = (ts.volMmk - 1)/lc.numthread.x + 1;
        // if (lc.numRegStorage > MAX_REG_STORAGE) {
        //   // Find number of threads that works
        //   lc.numthread.x = (( (ts.volMmk - 1)/MAX_REG_STORAGE)/prop.warpSize + 1)*prop.warpSize;
        //   lc.numRegStorage = (ts.volMmk - 1)/lc.numthread.x + 1;
        // }
      }

      lc.numRegStorage = bestNumRegStorage;
  */

      lc.numthread.x = ((ts.volMmk - 1)/(prop.warpSize*lc.numRegStorage) + 1)*prop.warpSize;
      lc.numthread.x = min(prop.maxThreadsPerBlock, lc.numthread.x);
      lc.numthread.y = 1;
      lc.numthread.z = 1;
      lc.numblock.x = max(1, ts.volMbar);
      lc.numblock.x = min(256, lc.numblock.x);
      lc.numblock.y = 1;
      lc.numblock.z = 1;

      lc.numRegStorage = (ts.volMmk - 1)/lc.numthread.x + 1;
      if (lc.numRegStorage > MAX_REG_STORAGE) {
        // Find number of threads that works
        lc.numthread.x = (( (ts.volMmk - 1)/MAX_REG_STORAGE)/prop.warpSize + 1)*prop.warpSize;
        lc.numRegStorage = (ts.volMmk - 1)/lc.numthread.x + 1;
      }

      // Check that we're not using too many threads or register storage
      //if (lc.numthread.x > prop.maxThreadsPerBlock || lc.numRegStorage > MAX_REG_STORAGE) return 0;

    }
    break;
    case cuttPlan_t::TiledSingleRank:
    {
      lc.numthread.x = TILEDIM;
      lc.numthread.y = TILEROWS;
      lc.numthread.z = 1;
      lc.numblock.x = (ts.volMm - 1)/TILEDIM + 1;
      lc.numblock.y = (ts.volMk - 1)/TILEDIM + 1;
      lc.numblock.z = ts.volMbar;
      lc.numblock.z = min(64/(lc.numblock.x*lc.numblock.y), lc.numblock.z);
      lc.numblock.z = max(1, lc.numblock.z);
      lc.shmemsize = 0;
      lc.numRegStorage = 0;
    }
    break;
    case cuttPlan_t::TiledLeadVolSame:
    {
      lc.numthread.x = TILEDIM;
      lc.numthread.y = TILEROWS;
      lc.numthread.z = 1;
      lc.numblock.x = (ts.volMm - 1)/TILEDIM + 1;
      lc.numblock.y = (ts.volMkBar - 1)/TILEDIM + 1;
      lc.numblock.z = ts.volMbar;
      lc.numblock.z = min(64/(lc.numblock.x*lc.numblock.y), lc.numblock.z);
      lc.numblock.z = max(1, lc.numblock.z);
      lc.shmemsize = 0;
      lc.numRegStorage = 0;
    }
    break;
  }

  if (lc.numblock.x > prop.maxGridSize[0] ||
    lc.numblock.y > prop.maxGridSize[1] ||
    lc.numblock.z > prop.maxGridSize[2]) return 0;

  // Return the number of active blocks with these settings
  return getNumActiveBlock(method, sizeofType, lc);
}

bool cuttKernel(cuttPlan_t& plan, void* dataIn, void* dataOut) {

  LaunchConfig& lc = plan.launchConfig;
  TensorSplit& ts = plan.tensorSplit;

#if 0
  printf("numthread %d %d %d numblock %d %d %d shmemsize %d numRegStorage %d\n",
    lc.numthread.x, lc.numthread.y, lc.numthread.z,
    lc.numblock.x, lc.numblock.y, lc.numblock.z,
    lc.shmemsize, lc.numRegStorage);
#endif

  switch(plan.method) {
    case cuttPlan_t::General:
    {
      switch(lc.numRegStorage) {
#define CALL(TYPE, NREG) \
    transposeGeneral<TYPE, NREG> <<< lc.numblock, lc.numthread, lc.shmemsize, plan.stream >>> \
      (ts.volMm, ts.volMk, ts.volMmk, ts.volMbar, \
      ts.sizeMmk, ts.sizeMbar, \
      plan.Mmk, plan.Mbar, plan.Msh, (TYPE *)dataIn, (TYPE *)dataOut)
        case 1:
        if (plan.sizeofType == 4) CALL(float,  1);
        if (plan.sizeofType == 8) CALL(double, 1);
        break;
        case 2:
        if (plan.sizeofType == 4) CALL(float,  2);
        if (plan.sizeofType == 8) CALL(double, 2);
        break;
        case 3:
        if (plan.sizeofType == 4) CALL(float,  3);
        if (plan.sizeofType == 8) CALL(double, 3);
        break;
        case 4:
        if (plan.sizeofType == 4) CALL(float,  4);
        if (plan.sizeofType == 8) CALL(double, 4);
        break;
        case 5:
        if (plan.sizeofType == 4) CALL(float,  5);
        if (plan.sizeofType == 8) CALL(double, 5);
        break;
        case 6:
        if (plan.sizeofType == 4) CALL(float,  6);
        if (plan.sizeofType == 8) CALL(double, 6);
        break;
        case 7:
        if (plan.sizeofType == 4) CALL(float,  7);
        if (plan.sizeofType == 8) CALL(double, 7);
        break;
        case 8:
        if (plan.sizeofType == 4) CALL(float,  8);
        if (plan.sizeofType == 8) CALL(double, 8);
        break;
        default:
        printf("cuttKernel no template implemented for numRegStorage %d\n", lc.numRegStorage);
        return false;
#undef CALL
      }

    }
    break;

    case cuttPlan_t::TiledSingleRank:
    {
#define CALL(TYPE) \
      transposeTiledSingleRank<TYPE> <<< lc.numblock, lc.numthread, 0, plan.stream >>> \
      (ts.volMbar, ts.sizeMbar, plan.tiledVol, plan.cuDimMk, plan.cuDimMm, \
        plan.Mbar, (TYPE *)dataIn, (TYPE *)dataOut)

      // dim3 numthread(TILEDIM, TILEROWS, 1);
      // dim3 numblock((plan.volMm-1)/TILEDIM+1, (plan.volMk-1)/TILEDIM+1, plan.volMbar);
      // numblock.z = min(256, plan.volMbar);
      // numblock.z = min(65535, numblock.z);

      // printf("numthread %d %d %d numblock %d %d %d\n", numthread.x, numthread.y, numthread.z,
      //   numblock.x, numblock.y, numblock.z);

      if (plan.sizeofType == 4) CALL(float);
      if (plan.sizeofType == 8) CALL(double);
#undef CALL
    }
    break;

    case cuttPlan_t::TiledLeadVolSame:
    {
#define CALL(TYPE) \
      transposeTiledLeadVolSame<TYPE> <<< lc.numblock, lc.numthread, 0, plan.stream >>> \
      (ts.volMbar, ts.sizeMbar, plan.cuDimMk, plan.cuDimMm, plan.tiledVol, \
        plan.Mbar, (TYPE *)dataIn, (TYPE *)dataOut)

      // dim3 numthread(TILEDIM, TILEROWS, 1);
      // dim3 numblock((plan.readVol.x-1)/TILEDIM+1, (plan.readVol.y-1)/TILEDIM+1, plan.volMbar);
      // numblock.z = min(256, plan.volMbar);
      // numblock.z = min(65535, numblock.z);

      // printf("numthread %d %d %d numblock %d %d %d\n", numthread.x, numthread.y, numthread.z,
      //   numblock.x, numblock.y, numblock.z);

      if (plan.sizeofType == 4) CALL(float);
      if (plan.sizeofType == 8) CALL(double);
#undef CALL
    }
    break;

  }

  cudaCheck(hipGetLastError());
  return true;
}
